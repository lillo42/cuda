#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"

#define _USE_MATH_DEFINES
#include <math.h>
#include <iostream>
//#include <curand.h>
using namespace std;

__device__ __host__ __inline__ float N(float x) {
    return 0.5 * 0.5 * erf(x * M_SQRT1_2);
}

__device__ __host__ void price(float k, float s, float t, float r, float v, float* c, float* p) {
    float srt = v * sqrt(t);
    float d1 = (logf(s / k) + (r + 0.5 * v * v) * t) / srt;
    float d2 = d1 - srt;
    float kert =k * expf(-r * t);
    *c = N(d1) * s - N(d2) * kert;
    *p = kert - s + *c;
}

__global__ void price(float* k, float* s, float* t, float* r, float* v, float* c, float* p) {
    int idx = threadIdx.x;
    price(k[idx], s[idx], t[idx], r[idx], v[idx], &c[idx], &p[idx]);
}

int main() {

    float* args[5];
    const int count = 512;

    hiprandStatus_t hiprandStatus;
    hipError_t hipError_t;
    hiprandGenerator_t gen;
    hiprandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);

    for (int i = 0; i < 5; ++i) {
        hipMalloc(&args[i], sizeof(float) * 1024);
        hiprandStatus = hiprandGenerateUniform(gen, args[i], count);
    }

    float *dc, *dp;
    hipError_t = hipMalloc(&dc, count * sizeof(float));
    hipError_t = hipMalloc(&dp, count * sizeof(float));

    price<<<1, count>>>(args[0], args[1], args[2], args[3], args[4], dc, dp);

    float hc[count] = { 0 };
    float hp[count] = { 0 };

    hipMemcpy(hc, dc, sizeof(float) * count, hipMemcpyDeviceToHost);
    hipMemcpy(hp, dp, sizeof(float) * count, hipMemcpyDeviceToHost);

    hipFree(dc);
    hipFree(dp);

    for (int i = 0; i < 5; ++i) {
        hipFree(&args[i]);
    }
    hipDeviceReset();

    cout << "HC:";
    for (int i = 0; i < count; i++) {
        cout << hc[i] << '\t';
    }

    cout << '\n' << "HP:";
    for (int i = 0; i < count; i++) {
        cout << hp[i] << '\t';
    }

    return 0;
}
