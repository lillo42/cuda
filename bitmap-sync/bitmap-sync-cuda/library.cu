#include "hip/hip_runtime.h"


#define _USE_MATH_DEFINES
#include <math.h>

__global__ void kernel(unsigned char* src) {
    __shared__ float temp[16][16];

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    const float period = 128.0f;

    temp[threadIdx.x][threadIdx.y] = 255 *
            (sinf(x * 2.0f * M_PI/period) + 1.0f) *
            (sinf(y * 2.0f * M_PI/period) + 1.0f) / 4.0f;

    __syncthreads();

    src[offset * 4] = 0;
    src[offset * 4 + 1] = temp[15 - threadIdx.x][15 - threadIdx.y];
    src[offset * 4 + 2] = 0;
    src[offset * 4 + 3] = 255;

}

extern "C" __declspec(dllexport) void GenerateBitmap(unsigned char* dst, int dimension) {
    int size = dimension * dimension * 4;
    hipError_t  status;

    unsigned char* src;
    status = hipMalloc(&src, size);

    dim3 blocks(dimension / 16, dimension / 16);
    dim3 threads(16, 16);

    kernel<<<blocks, threads>>>(src);

    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);

    hipFree(src);
}